#include "hip/hip_runtime.h"
#ifndef _BCDATA_CPP_
#define _BCDATA_CPP_

#include "BCDATA.h"

BC::BC(poly* opt_in, uint8_t numCons_in, poly* cons_in, uint8_t numEqus_in, poly* equs_in) {
	opt = opt_in;
	cons = cons_in;
	numCons = numCons_in;
	equs = equs_in;
	numEqus = numEqus_in;
	numUnit = 1;
	apex_numUnit = 1;
	numDimension = opt_in->numDimension;

	if (numDimension == 2) {
		MAX_UNIT_NUM = twod_MAX_UNIT_NUM;
	}
	else if (numDimension == 3) {
		MAX_UNIT_NUM = threed_MAX_UNIT_NUM;
	}
	else {
		MAX_UNIT_NUM = fourd_MAX_UNIT_NUM;
	}

	opt_degree = new uint8_t[numDimension];
	if (numCons > 0) con_degree = new uint8_t[numDimension];
	else con_degree = nullptr;
	if (numEqus > 0) equ_degree = new uint8_t[numDimension];
	else equ_degree = nullptr;

	opt_unitLength = con_unitLength = equ_unitLength = 1;
	for (uint8_t i = 0; i < numDimension; i++)
	{
		opt_degree[i] = opt_in->maxDegree[i];
		opt_unitLength *= opt_degree[i];

		if (numCons > 0) {
			con_degree[i] = 0;
			for (uint8_t j = 0; j < numCons; j++) {
				if (cons_in[j].maxDegree[i] > con_degree[i]) {
					con_degree[i] = cons_in[j].maxDegree[i];
				}
			}
			con_unitLength *= con_degree[i];
		}

		if (numEqus > 0) {
			equ_degree[i] = 0;
			for (uint8_t j = 0; j < numEqus; j++) {
				if (equs_in[j].maxDegree[i] > equ_degree[i]) {
					equ_degree[i] = equs_in[j].maxDegree[i];
				}
			}
			equ_unitLength *= equ_degree[i];
		}
	}

	//opt_BC = new float[MAX_UNIT_NUM * opt_unitLength];
	opt_BC = new float[opt_unitLength];
	memset(opt_BC, 0, opt_unitLength * sizeof(float));

	uint16_t index, currentPos, currentDegree;
	float BCterm;

	for (uint16_t i = 0; i < opt_in->numTerms; i++) {
		for (uint16_t j = 0; j < opt_unitLength; j++) {
			index = j;
			BCterm = 1;
			for (int k = 0; k < numDimension; k++) {
				currentDegree = index % opt_degree[k];
				currentPos = i * numDimension + k;
				BCterm *= choosenk[currentDegree][opt_in->degree[currentPos]] / choosenk[opt_degree[k] - 1][opt_in->degree[currentPos]];
				index = (index - currentDegree) / opt_degree[k];
			}
			opt_BC[j] += opt_in->coeff[i] * BCterm;
		}
	}

	hipMalloc((void**)&dev_opt_BC, MAX_UNIT_NUM * opt_unitLength * sizeof(float));
	hipMemcpy(dev_opt_BC, opt_BC, opt_unitLength * sizeof(float), hipMemcpyHostToDevice);

	pdValue = new float[numDimension];
	//pd_BC = new float[MAX_UNIT_NUM * numDimension * opt_unitLength];
	pd_BC = new float[numDimension * opt_unitLength];
	memset(pd_BC, 0, numDimension * opt_unitLength * sizeof(float));

	uint16_t pdOffset = 0;
	poly* pd = nullptr;
	float pdValueBuf = 0;
	for (uint8_t pdID = 0; pdID < numDimension; pdID++) {
		opt->partialDerivative(pd, pdValueBuf, pdID);
		for (uint16_t i = 0; i < pd->numTerms; i++) {
			for (uint16_t j = 0; j < opt_unitLength; j++) {
				index = j;
				BCterm = 1;
				for (int k = 0; k < numDimension; k++) {
					currentDegree = index % opt_degree[k];
					currentPos = i * numDimension + k;
					BCterm *= choosenk[currentDegree][pd->degree[currentPos]] / choosenk[opt_degree[k] - 1][pd->degree[currentPos]];
					index = (index - currentDegree) / opt_degree[k];
				}
				pd_BC[pdOffset + j] += pd->coeff[i] * BCterm;
			}
		}
		pdOffset += opt_unitLength;
		delete pd;

		pdValue[pdID] = pdValueBuf;
	}

	hipMalloc((void**)&dev_pd_BC, numDimension * MAX_UNIT_NUM * opt_unitLength * sizeof(float));
	hipMemcpy(dev_pd_BC, pd_BC, numDimension * opt_unitLength * sizeof(float), hipMemcpyHostToDevice);

	hipMalloc((void**)&dev_pdValue, numDimension * MAX_UNIT_NUM * sizeof(float));
	hipMemcpy(dev_pdValue, pdValue, numDimension * sizeof(float), hipMemcpyHostToDevice);

	if (numCons > 0) {
		//con_BC = new float[numCons * MAX_UNIT_NUM * con_unitLength];
		con_BC = new float[numCons * con_unitLength];
		memset(con_BC, 0, numCons * con_unitLength * sizeof(float));

		uint16_t conOffset = 0;

		for (uint8_t conID = 0; conID < numCons; conID++) {
			for (uint16_t i = 0; i < cons[conID].numTerms; i++) {
				for (uint16_t j = 0; j < con_unitLength; j++) {
					index = j;
					BCterm = 1;
					for (int k = 0; k < numDimension; k++) {
						currentDegree = index % con_degree[k];
						currentPos = i * numDimension + k;
						BCterm *= choosenk[currentDegree][cons[conID].degree[currentPos]] / choosenk[con_degree[k] - 1][cons[conID].degree[currentPos]];
						index = (index - currentDegree) / con_degree[k];
					}
					con_BC[conOffset + j] += cons[conID].coeff[i] * BCterm;
				}
			}
			conOffset += con_unitLength;
		}

		hipMalloc((void**)&dev_con_BC, numCons * MAX_UNIT_NUM * con_unitLength * sizeof(float));
		hipMemcpy(dev_con_BC, con_BC, numCons * con_unitLength * sizeof(float), hipMemcpyHostToDevice);
	}
	else {
		con_BC = nullptr;
		dev_con_BC = nullptr;
	}

	if (numEqus > 0) {
		//equ_BC = new float[numEqus * MAX_UNIT_NUM * equ_unitLength];
		equ_BC = new float[numEqus * equ_unitLength];
		memset(equ_BC, 0, numEqus * equ_unitLength * sizeof(float));

		uint16_t equOffset = 0;

		for (uint8_t equID = 0; equID < numEqus; equID++) {
			for (uint16_t i = 0; i < equs[equID].numTerms; i++) {
				for (uint16_t j = 0; j < equ_unitLength; j++) {
					index = j;
					BCterm = 1;
					for (int k = 0; k < numDimension; k++) {
						currentDegree = index % equ_degree[k];
						currentPos = i * numDimension + k;
						BCterm *= choosenk[currentDegree][equs[equID].degree[currentPos]] / choosenk[equ_degree[k] - 1][equs[equID].degree[currentPos]];
						index = (index - currentDegree) / equ_degree[k];
					}
					equ_BC[equOffset + j] += equs[equID].coeff[i] * BCterm;
				}
			}
			equOffset += equ_unitLength;
		}

		hipMalloc((void**)&dev_equ_BC, numEqus * MAX_UNIT_NUM * equ_unitLength * sizeof(float));
		hipMemcpy(dev_equ_BC, equ_BC, numEqus * equ_unitLength * sizeof(float), hipMemcpyHostToDevice);
	}
	else {
		equ_BC = nullptr;
		dev_equ_BC = nullptr;
	}

	//interval = new uint32_t[MAX_UNIT_NUM * numDimension];
	interval = new uint32_t[numDimension];
	memset(interval, 0, numDimension * sizeof(uint32_t));
	hipMalloc((void**)&dev_interval, MAX_UNIT_NUM * numDimension * sizeof(uint32_t));
	hipMemcpy(dev_interval, interval, numDimension * sizeof(uint32_t), hipMemcpyHostToDevice);

	int_iter = new uint8_t[numDimension];
	memset(int_iter, 0, numDimension * sizeof(uint8_t));

	bdMin = new float[MAX_UNIT_NUM];
	hipMalloc((void**)&dev_bdMin, MAX_UNIT_NUM * sizeof(float));

	bdMax = new float[MAX_UNIT_NUM];
	hipMalloc((void**)&dev_bdMax, MAX_UNIT_NUM * sizeof(float));

	//pdFlag = new bool[MAX_UNIT_NUM * numDimension];
	pdFlag = new bool[numDimension];
	for (uint8_t i = 0; i < numDimension; i++) {
		pdFlag[i] = true;
	}
	hipMalloc((void**)&dev_pdFlag, numDimension * MAX_UNIT_NUM * sizeof(bool));
	hipMemcpy(dev_pdFlag, pdFlag, numDimension * sizeof(bool), hipMemcpyHostToDevice);

	dFlag = new bool[MAX_UNIT_NUM];
	hipMalloc((void**)&dev_dFlag, MAX_UNIT_NUM * sizeof(bool));

	if (numCons > 0) {
		//consFlag = new char[numCons * MAX_UNIT_NUM];
		consFlag = new char[numCons];
		for (uint8_t i = 0; i < numCons; i++) {
			consFlag[i] = 1;
		}
		hipMalloc((void**)&dev_consFlag, numCons * MAX_UNIT_NUM * sizeof(char));
		hipMemcpy(dev_consFlag, consFlag, numCons * sizeof(char), hipMemcpyHostToDevice);

		intFlag = new char[MAX_UNIT_NUM];
		hipMalloc((void**)&dev_intFlag, MAX_UNIT_NUM * sizeof(char));
	}
	else {
		consFlag = nullptr;
		dev_consFlag = nullptr;
		intFlag = nullptr;
		dev_intFlag = nullptr;
	}

	if (numEqus > 0) {
		//equsFlag = new bool[numEqus * MAX_UNIT_NUM];
		equsFlag = new bool[numEqus];
		hipMalloc((void**)&dev_equsFlag, numEqus * MAX_UNIT_NUM * sizeof(bool));

		eFlag = new bool[MAX_UNIT_NUM];
		hipMalloc((void**)&dev_eFlag, MAX_UNIT_NUM * sizeof(bool));
	}
	else {
		equsFlag = nullptr;
		dev_equsFlag = nullptr;
		eFlag = nullptr;
		dev_eFlag = nullptr;
	}


	elimPos = new uint32_t[MAX_UNIT_NUM];
	hipMalloc((void**)&dev_elimPos, MAX_UNIT_NUM * sizeof(uint32_t));

	savePos = new uint32_t[MAX_UNIT_NUM];
	hipMalloc((void**)&dev_savePos, MAX_UNIT_NUM * sizeof(uint32_t));

	intervalRes = nullptr;
	dev_intervalRes = nullptr;

	candidates = nullptr;

	//debug = new float[MAX_UNIT_NUM * numCons * 2];
	//hipMalloc((void**)&dev_debug, MAX_UNIT_NUM * numCons * 2 * sizeof(float));

	final_index = 0;

	final_result = nullptr;
}

BC::~BC() {
	delete[] opt_degree;

	if (numCons > 0) delete[] con_degree;

	if (numEqus > 0) delete[] equ_degree;

	delete[] opt_BC;
	hipFree(dev_opt_BC);

	delete[] pd_BC;
	hipFree(dev_pd_BC);

	delete[] pdValue;
	hipFree(dev_pdValue);

	if (numCons > 0) {
		delete[] con_BC;
		hipFree(dev_con_BC);
	}

	if (numEqus > 0) {
		delete[] equ_BC;
		hipFree(dev_equ_BC);
	}

	delete[] interval;
	hipFree(dev_interval);

	delete[] int_iter;

	delete[] bdMin;
	hipFree(dev_bdMin);

	delete[] bdMax;
	hipFree(dev_bdMax);

	delete[] pdFlag;
	hipFree(dev_pdFlag);

	delete[] dFlag;
	hipFree(dev_dFlag);

	if (numCons > 0) {
		delete[] consFlag;
		hipFree(dev_consFlag);

		delete[] intFlag;
		hipFree(dev_intFlag);
	}

	if (numEqus > 0) {
		delete[] equsFlag;
		hipFree(dev_equsFlag);

		delete[] eFlag;
		hipFree(dev_eFlag);
	}

	delete[] elimPos;
	hipFree(dev_elimPos);

	delete[] savePos;
	hipFree(dev_savePos);

	if (intervalRes != nullptr) {
		delete[] intervalRes;
		hipFree(dev_intervalRes);
	}

	if (candidates != nullptr) delete[] candidates;

	if (final_result != nullptr) delete[] final_result;

	//delete[] debug;
	//hipFree(dev_debug);
}

void BC::debug_print()
{
	int debugNumUnit = numUnit > 10 ? 10 : numUnit;
	/*
	mexPrintf("PRINT OPT BC\n");
	hipMemcpy(opt_BC, dev_opt_BC, debugNumUnit * opt_unitLength * sizeof(float), hipMemcpyDeviceToHost);
	for (uint32_t i = 0; i < debugNumUnit; i++)
	{
		mexPrintf("BC %d. \n", i);
		for (int j = 0; j < opt_unitLength; j++)
		{
			mexPrintf("%.6f ", opt_BC[i * opt_unitLength + j]);
		}
		mexPrintf("\n");
	}
	mexPrintf("\n\n");
	
	mexPrintf("PRINT PD BC\n");
	hipMemcpy(pd_BC, dev_pd_BC, debugNumUnit * numDimension * opt_unitLength * sizeof(float), hipMemcpyDeviceToHost);
	for (uint32_t i = 0; i < debugNumUnit * numDimension; i++)
	{
		mexPrintf("BC %d. \n", i);
		for (int j = 0; j < opt_unitLength; j++)
		{
			mexPrintf("%.6f ", pd_BC[i * opt_unitLength + j]);
		}
		mexPrintf("\n");
	}
	mexPrintf("\n\n");
	
	if (numCons > 0) {
		mexPrintf("PRINT CON BC\n");
		hipMemcpy(con_BC, dev_con_BC, debugNumUnit * numCons * con_unitLength * sizeof(float), hipMemcpyDeviceToHost);
		for (uint32_t i = 0; i < debugNumUnit * numCons; i++)
		{
			mexPrintf("BC %d. \n", i);
			for (int j = 0; j < con_unitLength; j++)
			{
				mexPrintf("%.6f ", con_BC[i * con_unitLength + j]);
			}
			mexPrintf("\n");
		}
		mexPrintf("\n\n");
	}
	
	if (numEqus > 0) {
		mexPrintf("PRINT EQU BC\n");
		hipMemcpy(equ_BC, dev_equ_BC, debugNumUnit * numEqus * equ_unitLength * sizeof(float), hipMemcpyDeviceToHost);
		for (uint32_t i = 0; i < debugNumUnit * numEqus; i++)
		{
			mexPrintf("BC %d. \n", i);
			for (int j = 0; j < equ_unitLength; j++)
			{
				mexPrintf("%.6f ", equ_BC[i * equ_unitLength + j]);
			}
			mexPrintf("\n");
		}
		mexPrintf("\n\n");
	}
	*/
	if (numCons > 0) {
		hipMemcpy(consFlag, dev_consFlag, debugNumUnit * numCons * sizeof(char), hipMemcpyDeviceToHost);
		hipMemcpy(intFlag, dev_intFlag, debugNumUnit * sizeof(char), hipMemcpyDeviceToHost);
	}
	if (numEqus > 0) {
		hipMemcpy(equsFlag, dev_equsFlag, debugNumUnit * numEqus * sizeof(bool), hipMemcpyDeviceToHost);
		hipMemcpy(eFlag, dev_eFlag, debugNumUnit * sizeof(bool), hipMemcpyDeviceToHost);
	}
	hipMemcpy(pdFlag, dev_pdFlag, debugNumUnit * numDimension * sizeof(bool), hipMemcpyDeviceToHost);
	hipMemcpy(dFlag, dev_dFlag, debugNumUnit * sizeof(bool), hipMemcpyDeviceToHost);
	hipMemcpy(interval, dev_interval, debugNumUnit * numDimension * sizeof(uint32_t), hipMemcpyDeviceToHost);
	hipMemcpy(bdMin, dev_bdMin, debugNumUnit * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(bdMax, dev_bdMax, debugNumUnit * sizeof(float), hipMemcpyDeviceToHost);

	hipMemcpy(debug, dev_debug, debugNumUnit * numCons * 2 * sizeof(float), hipMemcpyDeviceToHost);

	mexPrintf("PRINT UNIT INFO\n");
	for (uint32_t i = 0; i < debugNumUnit; i++)
	{
		mexPrintf("%d. [", i);
		for (uint8_t j = 0; j < numDimension; j++) {
			mexPrintf(" %d", interval[i * numDimension + j]);
		}
		mexPrintf("]  ");

		mexPrintf("[%.6f %.6f]  | ", bdMin[i], bdMax[i]);

		if (numCons > 0) {
			for (uint8_t j = 0; j < numCons; j++) {
				mexPrintf("%d ", consFlag[i * numCons + j]);
			}
			mexPrintf(":%d | ", intFlag[i]);
		}

		if (numEqus > 0) {
			for (uint8_t j = 0; j < numEqus; j++) {
				mexPrintf("%d ", equsFlag[i * numEqus + j]);
			}
			mexPrintf(":%d | ", eFlag[i]);
		}

		for (uint8_t j = 0; j < numDimension; j++) {
			mexPrintf("%d ", pdFlag[i * numDimension + j]);
		}
		mexPrintf(":%d\n", dFlag[i]);
	}
	mexPrintf("\n");
}

void BC::dilation(uint8_t dim) {
	if (numDimension == 2) {
		if (dim == 0) {
			biBCdilationKernelPart1Forx1 << < numUnit, opt_degree[1] >> > (dev_opt_BC, opt_degree[0], opt_unitLength);
			biBCdilationKernelPart1Forx1 << < numUnit * numDimension, opt_degree[1] >> > (dev_pd_BC, opt_degree[0], opt_unitLength);
			if (numCons > 0) biBCdilationKernelPart1Forx1 << < numUnit * numCons, con_degree[1] >> > (dev_con_BC, con_degree[0], con_unitLength);
			if (numEqus > 0) biBCdilationKernelPart1Forx1 << < numUnit * numEqus, equ_degree[1] >> > (dev_equ_BC, equ_degree[0], equ_unitLength);
		}
		else {
			biBCdilationKernelPart1Forx2 << < numUnit, opt_degree[0] >> > (dev_opt_BC, opt_degree[1], opt_unitLength);
			biBCdilationKernelPart1Forx2 << < numUnit * numDimension, opt_degree[0] >> > (dev_pd_BC, opt_degree[1], opt_unitLength);
			if (numCons > 0) biBCdilationKernelPart1Forx2 << < numUnit * numCons, con_degree[0] >> > (dev_con_BC, con_degree[1], con_unitLength);
			if (numEqus > 0) biBCdilationKernelPart1Forx2 << < numUnit * numEqus, equ_degree[0] >> > (dev_equ_BC, equ_degree[1], equ_unitLength);
		}
	}
	else if (numDimension == 3) {
		if (dim == 0) {
			dim3 block1(opt_degree[1], opt_degree[2], 1);
			triBCdilationKernelPart1Forx1 << < numUnit, block1 >> > (dev_opt_BC, opt_degree[0], opt_unitLength);
			triBCdilationKernelPart1Forx1 << < numUnit * numDimension, block1 >> > (dev_pd_BC, opt_degree[0], opt_unitLength);

			if (numCons > 0) {
				dim3 block2(con_degree[1], con_degree[2], 1);
				triBCdilationKernelPart1Forx1 << < numUnit * numCons, block2 >> > (dev_con_BC, con_degree[0], con_unitLength);
			}

			if (numEqus > 0) {
				dim3 block3(equ_degree[1], equ_degree[2], 1);
				triBCdilationKernelPart1Forx1 << < numUnit * numEqus, block3 >> > (dev_equ_BC, equ_degree[0], equ_unitLength);
			}
		}
		else if (dim == 1) {
			dim3 block1(opt_degree[0], opt_degree[2], 1);
			triBCdilationKernelPart1Forx2 << < numUnit, block1 >> > (dev_opt_BC, opt_degree[1], opt_unitLength);
			triBCdilationKernelPart1Forx2 << < numUnit * numDimension, block1 >> > (dev_pd_BC, opt_degree[1], opt_unitLength);

			if (numCons > 0) {
				dim3 block2(con_degree[0], con_degree[2], 1);
				triBCdilationKernelPart1Forx2 << < numUnit * numCons, block2 >> > (dev_con_BC, con_degree[1], con_unitLength);
			}

			if (numEqus > 0) {
				dim3 block3(equ_degree[0], equ_degree[2], 1);
				triBCdilationKernelPart1Forx2 << < numUnit * numEqus, block3 >> > (dev_equ_BC, equ_degree[1], equ_unitLength);
			}
		}
		else {
			dim3 block1(opt_degree[0], opt_degree[1], 1);
			triBCdilationKernelPart1Forx3 << < numUnit, block1 >> > (dev_opt_BC, opt_degree[2], opt_unitLength);
			triBCdilationKernelPart1Forx3 << < numUnit * numDimension, block1 >> > (dev_pd_BC, opt_degree[2], opt_unitLength);

			if (numCons > 0) {
				dim3 block2(con_degree[0], con_degree[1], 1);
				triBCdilationKernelPart1Forx3 << < numUnit * numCons, block2 >> > (dev_con_BC, con_degree[2], con_unitLength);
			}

			if (numEqus > 0) {
				dim3 block3(equ_degree[0], equ_degree[1], 1);
				triBCdilationKernelPart1Forx3 << < numUnit * numEqus, block3 >> > (dev_equ_BC, equ_degree[2], equ_unitLength);
			}
		}
	}
	else {
		if (dim == 0) {
			dim3 block1(opt_degree[1], opt_degree[2], opt_degree[3]);
			quadBCdilationKernelPart1Forx1 << < numUnit, block1 >> > (dev_opt_BC, opt_degree[0], opt_unitLength);
			quadBCdilationKernelPart1Forx1 << < numUnit * numDimension, block1 >> > (dev_pd_BC, opt_degree[0], opt_unitLength);

			if (numCons > 0) {
				dim3 block2(con_degree[1], con_degree[2], con_degree[3]);
				quadBCdilationKernelPart1Forx1 << < numUnit * numCons, block2 >> > (dev_con_BC, con_degree[0], con_unitLength);
			}

			if (numEqus > 0) {
				dim3 block3(equ_degree[1], equ_degree[2], equ_degree[3]);
				quadBCdilationKernelPart1Forx1 << < numUnit * numEqus, block3 >> > (dev_equ_BC, equ_degree[0], equ_unitLength);
			}
		}
		else if (dim == 1) {
			dim3 block1(opt_degree[0], opt_degree[2], opt_degree[3]);
			quadBCdilationKernelPart1Forx2 << < numUnit, block1 >> > (dev_opt_BC, opt_degree[1], opt_unitLength);
			quadBCdilationKernelPart1Forx2 << < numUnit * numDimension, block1 >> > (dev_pd_BC, opt_degree[1], opt_unitLength);

			if (numCons > 0) {
				dim3 block2(con_degree[0], con_degree[2], con_degree[3]);
				quadBCdilationKernelPart1Forx2 << < numUnit * numCons, block2 >> > (dev_con_BC, con_degree[1], con_unitLength);
			}

			if (numEqus > 0) {
				dim3 block3(equ_degree[0], equ_degree[2], equ_degree[3]);
				quadBCdilationKernelPart1Forx2 << < numUnit * numEqus, block3 >> > (dev_equ_BC, equ_degree[1], equ_unitLength);
			}
		}
		else if (dim == 2) {
			dim3 block1(opt_degree[0], opt_degree[1], opt_degree[3]);
			quadBCdilationKernelPart1Forx3 << < numUnit, block1 >> > (dev_opt_BC, opt_degree[2], opt_unitLength);
			quadBCdilationKernelPart1Forx3 << < numUnit * numDimension, block1 >> > (dev_pd_BC, opt_degree[2], opt_unitLength);

			if (numCons > 0) {
				dim3 block2(con_degree[0], con_degree[1], con_degree[3]);
				quadBCdilationKernelPart1Forx3 << < numUnit * numCons, block2 >> > (dev_con_BC, con_degree[2], con_unitLength);
			}

			if (numEqus > 0) {
				dim3 block3(equ_degree[0], equ_degree[1], equ_degree[3]);
				quadBCdilationKernelPart1Forx3 << < numUnit * numEqus, block3 >> > (dev_equ_BC, equ_degree[2], equ_unitLength);
			}
		}
		else {
			dim3 block1(opt_degree[0], opt_degree[1], opt_degree[2]);
			quadBCdilationKernelPart1Forx4 << < numUnit, block1 >> > (dev_opt_BC, opt_degree[3], opt_unitLength);
			quadBCdilationKernelPart1Forx4 << < numUnit * numDimension, block1 >> > (dev_pd_BC, opt_degree[3], opt_unitLength);

			if (numCons > 0) {
				dim3 block2(con_degree[0], con_degree[1], con_degree[2]);
				quadBCdilationKernelPart1Forx4 << < numUnit * numCons, block2 >> > (dev_con_BC, con_degree[3], con_unitLength);
			}

			if (numEqus > 0) {
				dim3 block3(equ_degree[0], equ_degree[1], equ_degree[2]);
				quadBCdilationKernelPart1Forx4 << < numUnit * numEqus, block3 >> > (dev_equ_BC, equ_degree[3], equ_unitLength);
			}
		}
	}

	BCdilationKernelPart2 << < numUnit, numDimension >> > (dev_interval, dev_pdFlag, dim, dev_pdValue);

	BCdilationKernelPart3 << < numUnit, numCons >> > (dev_consFlag);

	numUnit <<= 1;
}

__global__ void biBCdilationKernelPart1Forx1(float* target_BC, uint8_t x1degree, uint16_t unitLength)
{
	int unitID = blockIdx.x;
	int unitOffset = gridDim.x;
	int x2ID = threadIdx.x;
	int BCLeftBase = unitID * unitLength + x2ID * x1degree + x1degree - 1;
	int BCLeftGrow = (unitID + unitOffset) * unitLength + x2ID * x1degree + x1degree - 1;
	int BCRightBase = unitID * unitLength + x2ID * x1degree;
	float BCterm;
	for (int x1pos = 0; x1pos < x1degree; x1pos++) {
		BCterm = 0;

		for (int addID = 0; addID <= x1pos; addID++) {
			BCterm += target_BC[BCLeftBase - addID] * dilationMat[x1pos][addID];
		}

		target_BC[BCLeftGrow - x1pos] = BCterm;
	}

	for (int x1pos = x1degree - 1; x1pos >= 0; x1pos--) {
		BCterm = 0;

		for (int addID = 0; addID <= x1pos; addID++) {
			BCterm += target_BC[BCRightBase + addID] * dilationMat[x1pos][addID];
		}

		target_BC[BCRightBase + x1pos] = BCterm;
	}
}

__global__ void biBCdilationKernelPart1Forx2(float* target_BC, uint8_t x2degree, uint16_t unitLength)
{
	int unitID = blockIdx.x;
	int unitOffset = gridDim.x;
	int x1ID = threadIdx.x;
	int x1degree = blockDim.x;
	int BCLeftBase = unitID * unitLength + (x2degree - 1) * x1degree + x1ID;
	int BCLeftGrow = (unitID + unitOffset) * unitLength + (x2degree - 1) * x1degree + x1ID;
	int BCRightBase = unitID * unitLength + x1ID;
	float BCterm;
	for (int x2pos = 0; x2pos < x2degree; x2pos++) {
		BCterm = 0;

		for (int addID = 0; addID <= x2pos; addID++) {
			BCterm += target_BC[BCLeftBase - addID * x1degree] * dilationMat[x2pos][addID];
		}

		target_BC[BCLeftGrow - x2pos * x1degree] = BCterm;
	}

	for (int x2pos = x2degree - 1; x2pos >= 0; x2pos--) {
		BCterm = 0;

		for (int addID = 0; addID <= x2pos; addID++) {
			BCterm += target_BC[BCRightBase + addID * x1degree] * dilationMat[x2pos][addID];
		}

		target_BC[BCRightBase + x2pos * x1degree] = BCterm;
	}
}

__global__ void triBCdilationKernelPart1Forx1(float* target_BC, uint8_t x1degree, uint16_t unitLength)
{
	int unitID = blockIdx.x;
	int unitOffset = gridDim.x;
	int x2ID = threadIdx.x;
	int x3ID = threadIdx.y;
	int x2degreePos = x1degree;
	int x3degreePos = blockDim.x * x2degreePos;
	int BCLeftBase = unitID * unitLength + x3ID * x3degreePos + x2ID * x2degreePos + x1degree - 1;
	int BCLeftGrow = (unitID + unitOffset) * unitLength + x3ID * x3degreePos + x2ID * x2degreePos + x1degree - 1;
	int BCRightBase = unitID * unitLength + x3ID * x3degreePos + x2ID * x2degreePos;
	float BCterm;
	for (int x1pos = 0; x1pos < x1degree; x1pos++) {
		BCterm = 0;

		for (int addID = 0; addID <= x1pos; addID++) {
			BCterm += target_BC[BCLeftBase - addID] * dilationMat[x1pos][addID];
		}

		target_BC[BCLeftGrow - x1pos] = BCterm;
	}

	for (int x1pos = x1degree - 1; x1pos >= 0; x1pos--) {
		BCterm = 0;

		for (int addID = 0; addID <= x1pos; addID++) {
			BCterm += target_BC[BCRightBase + addID] * dilationMat[x1pos][addID];
		}

		target_BC[BCRightBase + x1pos] = BCterm;
	}
}

__global__ void triBCdilationKernelPart1Forx2(float* target_BC, uint8_t x2degree, uint16_t unitLength)
{
	int unitID = blockIdx.x;
	int unitOffset = gridDim.x;
	int x1ID = threadIdx.x;
	int x3ID = threadIdx.y;
	int x2degreePos = blockDim.x;
	int x3degreePos = x2degree * x2degreePos;
	int BCLeftBase = unitID * unitLength + x3ID * x3degreePos + (x2degree - 1) * x2degreePos + x1ID;
	int BCLeftGrow = (unitID + unitOffset) * unitLength + x3ID * x3degreePos + (x2degree - 1) * x2degreePos + x1ID;
	int BCRightBase = unitID * unitLength + x3ID * x3degreePos + x1ID;
	float BCterm;
	for (int x2pos = 0; x2pos < x2degree; x2pos++) {
		BCterm = 0;

		for (int addID = 0; addID <= x2pos; addID++) {
			BCterm += target_BC[BCLeftBase - addID * x2degreePos] * dilationMat[x2pos][addID];
		}

		target_BC[BCLeftGrow - x2pos * x2degreePos] = BCterm;
	}

	for (int x2pos = x2degree - 1; x2pos >= 0; x2pos--) {
		BCterm = 0;

		for (int addID = 0; addID <= x2pos; addID++) {
			BCterm += target_BC[BCRightBase + addID * x2degreePos] * dilationMat[x2pos][addID];
		}

		target_BC[BCRightBase + x2pos * x2degreePos] = BCterm;
	}
}

__global__ void triBCdilationKernelPart1Forx3(float* target_BC, uint8_t x3degree, uint16_t unitLength)
{
	int unitID = blockIdx.x;
	int unitOffset = gridDim.x;
	int x1ID = threadIdx.x;
	int x2ID = threadIdx.y;
	int x2degreePos = blockDim.x;
	int x3degreePos = blockDim.y * x2degreePos;
	int BCLeftBase = unitID * unitLength + (x3degree - 1) * x3degreePos + x2ID * x2degreePos + x1ID;
	int BCLeftGrow = (unitID + unitOffset) * unitLength + (x3degree - 1) * x3degreePos + x2ID * x2degreePos + x1ID;
	int BCRightBase = unitID * unitLength + x2ID * x2degreePos + x1ID;
	float BCterm;
	for (int x3pos = 0; x3pos < x3degree; x3pos++) {
		BCterm = 0;

		for (int addID = 0; addID <= x3pos; addID++) {
			BCterm += target_BC[BCLeftBase - addID * x3degreePos] * dilationMat[x3pos][addID];
		}

		target_BC[BCLeftGrow - x3pos * x3degreePos] = BCterm;
	}

	for (int x3pos = x3degree - 1; x3pos >= 0; x3pos--) {
		BCterm = 0;

		for (int addID = 0; addID <= x3pos; addID++) {
			BCterm += target_BC[BCRightBase + addID * x3degreePos] * dilationMat[x3pos][addID];
		}

		target_BC[BCRightBase + x3pos * x3degreePos] = BCterm;
	}
}

__global__ void quadBCdilationKernelPart1Forx1(float* target_BC, uint8_t x1degree, uint16_t unitLength)
{
	int unitID = blockIdx.x;
	int unitOffset = gridDim.x;
	int x2ID = threadIdx.x;
	int x3ID = threadIdx.y;
	int x4ID = threadIdx.z;
	int x2degreePos = x1degree;
	int x3degreePos = blockDim.x * x2degreePos;
	int x4degreePos = blockDim.y * x3degreePos;
	int BCLeftBase = unitID * unitLength + x4ID * x4degreePos + x3ID * x3degreePos + x2ID * x2degreePos + x1degree - 1;
	int BCLeftGrow = (unitID + unitOffset) * unitLength + x4ID * x4degreePos + x3ID * x3degreePos + x2ID * x2degreePos + x1degree - 1;
	int BCRightBase = unitID * unitLength + x4ID * x4degreePos + x3ID * x3degreePos + x2ID * x2degreePos;
	float BCterm;
	for (int x1pos = 0; x1pos < x1degree; x1pos++) {
		BCterm = 0;

		for (int addID = 0; addID <= x1pos; addID++) {
			BCterm += target_BC[BCLeftBase - addID] * dilationMat[x1pos][addID];
		}

		target_BC[BCLeftGrow - x1pos] = BCterm;
	}

	for (int x1pos = x1degree - 1; x1pos >= 0; x1pos--) {
		BCterm = 0;

		for (int addID = 0; addID <= x1pos; addID++) {
			BCterm += target_BC[BCRightBase + addID] * dilationMat[x1pos][addID];
		}

		target_BC[BCRightBase + x1pos] = BCterm;
	}
}

__global__ void quadBCdilationKernelPart1Forx2(float* target_BC, uint8_t x2degree, uint16_t unitLength)
{
	int unitID = blockIdx.x;
	int unitOffset = gridDim.x;
	int x1ID = threadIdx.x;
	int x3ID = threadIdx.y;
	int x4ID = threadIdx.z;
	int x2degreePos = blockDim.x;
	int x3degreePos = x2degree * x2degreePos;
	int x4degreePos = blockDim.y * x3degreePos;
	int BCLeftBase = unitID * unitLength + x4ID * x4degreePos + x3ID * x3degreePos + (x2degree - 1) * x2degreePos + x1ID;
	int BCLeftGrow = (unitID + unitOffset) * unitLength + x4ID * x4degreePos + x3ID * x3degreePos + (x2degree - 1) * x2degreePos + x1ID;
	int BCRightBase = unitID * unitLength + x4ID * x4degreePos + x3ID * x3degreePos + x1ID;
	float BCterm;
	for (int x2pos = 0; x2pos < x2degree; x2pos++) {
		BCterm = 0;

		for (int addID = 0; addID <= x2pos; addID++) {
			BCterm += target_BC[BCLeftBase - addID * x2degreePos] * dilationMat[x2pos][addID];
		}

		target_BC[BCLeftGrow - x2pos * x2degreePos] = BCterm;
	}

	for (int x2pos = x2degree - 1; x2pos >= 0; x2pos--) {
		BCterm = 0;

		for (int addID = 0; addID <= x2pos; addID++) {
			BCterm += target_BC[BCRightBase + addID * x2degreePos] * dilationMat[x2pos][addID];
		}

		target_BC[BCRightBase + x2pos * x2degreePos] = BCterm;
	}
}

__global__ void quadBCdilationKernelPart1Forx3(float* target_BC, uint8_t x3degree, uint16_t unitLength)
{
	int unitID = blockIdx.x;
	int unitOffset = gridDim.x;
	int x1ID = threadIdx.x;
	int x2ID = threadIdx.y;
	int x4ID = threadIdx.z;
	int x2degreePos = blockDim.x;
	int x3degreePos = blockDim.y * x2degreePos;
	int x4degreePos = x3degree * x3degreePos;
	int BCLeftBase = unitID * unitLength + x4ID * x4degreePos + (x3degree - 1) * x3degreePos + x2ID * x2degreePos + x1ID;
	int BCLeftGrow = (unitID + unitOffset) * unitLength + x4ID * x4degreePos + (x3degree - 1) * x3degreePos + x2ID * x2degreePos + x1ID;
	int BCRightBase = unitID * unitLength + x4ID * x4degreePos + x2ID * x2degreePos + x1ID;
	float BCterm;
	for (int x3pos = 0; x3pos < x3degree; x3pos++) {
		BCterm = 0;

		for (int addID = 0; addID <= x3pos; addID++) {
			BCterm += target_BC[BCLeftBase - addID * x3degreePos] * dilationMat[x3pos][addID];
		}

		target_BC[BCLeftGrow - x3pos * x3degreePos] = BCterm;
	}

	for (int x3pos = x3degree - 1; x3pos >= 0; x3pos--) {
		BCterm = 0;

		for (int addID = 0; addID <= x3pos; addID++) {
			BCterm += target_BC[BCRightBase + addID * x3degreePos] * dilationMat[x3pos][addID];
		}

		target_BC[BCRightBase + x3pos * x3degreePos] = BCterm;
	}
}

__global__ void quadBCdilationKernelPart1Forx4(float* target_BC, uint8_t x4degree, uint16_t unitLength)
{
	int unitID = blockIdx.x;
	int unitOffset = gridDim.x;
	int x1ID = threadIdx.x;
	int x2ID = threadIdx.y;
	int x3ID = threadIdx.z;
	int x2degreePos = blockDim.x;
	int x3degreePos = blockDim.y * x2degreePos;
	int x4degreePos = blockDim.z * x3degreePos;
	int BCLeftBase = unitID * unitLength + (x4degree - 1) * x4degreePos + x3ID * x3degreePos + x2ID * x2degreePos + x1ID;
	int BCLeftGrow = (unitID + unitOffset) * unitLength + (x4degree - 1) * x4degreePos + x3ID * x3degreePos + x2ID * x2degreePos + x1ID;
	int BCRightBase = unitID * unitLength + x3ID * x3degreePos + x2ID * x2degreePos + x1ID;
	float BCterm;
	for (int x4pos = 0; x4pos < x4degree; x4pos++) {
		BCterm = 0;

		for (int addID = 0; addID <= x4pos; addID++) {
			BCterm += target_BC[BCLeftBase - addID * x4degreePos] * dilationMat[x4pos][addID];
		}

		target_BC[BCLeftGrow - x4pos * x4degreePos] = BCterm;
	}

	for (int x4pos = x4degree - 1; x4pos >= 0; x4pos--) {
		BCterm = 0;

		for (int addID = 0; addID <= x4pos; addID++) {
			BCterm += target_BC[BCRightBase + addID * x4degreePos] * dilationMat[x4pos][addID];
		}

		target_BC[BCRightBase + x4pos * x4degreePos] = BCterm;
	}
}

__global__ void BCdilationKernelPart2(uint32_t* target_interval, bool* target_pdFlag, uint8_t dim, float* target_pdValue) {
	int unitID = blockIdx.x;
	int intID = threadIdx.x;
	int numDimension = blockDim.x;
	int numUnit = gridDim.x;

	if (intID == dim) {
		target_interval[unitID * numDimension + intID] <<= 1;
		target_interval[(unitID + numUnit) * numDimension + intID] = target_interval[unitID * numDimension + intID] + 1;
	}
	else {
		target_interval[(unitID + numUnit) * numDimension + intID] = target_interval[unitID * numDimension + intID];
	}

	target_pdValue[(unitID + numUnit) * numDimension + intID] = target_pdValue[unitID * numDimension + intID];
	target_pdFlag[(unitID + numUnit) * numDimension + intID] = target_pdFlag[unitID * numDimension + intID];
}

__global__ void BCdilationKernelPart3(char* target_consFlag) {
	int unitID = blockIdx.x;
	int valueID = threadIdx.x;
	int numCons = blockDim.x;
	int numUnit = gridDim.x;
	target_consFlag[(unitID + numUnit) * numCons + valueID] = target_consFlag[unitID * numCons + valueID];
}

void BC::findFlag() {
	if (numCons > 0) {
		BCfindFlagKernel << < numUnit * numCons, con_unitLength >> > (dev_consFlag, dev_con_BC);
		BCfindIntFlagKernel << < numUnit, 1 >> > (dev_intFlag, dev_consFlag, numCons);

		hipMemcpy(intFlag, dev_intFlag, numUnit * sizeof(char), hipMemcpyDeviceToHost);
	}

	if (numEqus > 0) {
		BCfindEquFlagKernel << < numUnit * numEqus, equ_unitLength >> > (dev_equsFlag, dev_equ_BC);
		BCfindEFlagKernel << < numUnit, 1 >> > (dev_eFlag, dev_equsFlag, numEqus);

		hipMemcpy(eFlag, dev_eFlag, numUnit * sizeof(bool), hipMemcpyDeviceToHost);
	}

	BCfindBoundKernel << < numUnit, opt_unitLength >> > (dev_bdMin, dev_bdMax, dev_opt_BC);

	hipMemcpy(bdMin, dev_bdMin, numUnit * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(bdMax, dev_bdMax, numUnit * sizeof(float), hipMemcpyDeviceToHost);

	BCfindDerivativeKernel << < numUnit * numDimension, opt_unitLength >> > (dev_pdFlag, dev_pd_BC, dev_pdValue);
	
	if(numDimension == 2){
		biBCfinddFlagKernel << < numUnit, 1 >> > (dev_dFlag, dev_pdFlag, numDimension, dev_interval, int_iter[0], int_iter[1]);
	}
	else if(numDimension == 3){
		triBCfinddFlagKernel << < numUnit, 1 >> > (dev_dFlag, dev_pdFlag, numDimension, dev_interval, int_iter[0], int_iter[1], int_iter[2]);
	}
	else if(numDimension == 4){
		quadBCfinddFlagKernel << < numUnit, 1 >> > (dev_dFlag, dev_pdFlag, numDimension, dev_interval, int_iter[0], int_iter[1], int_iter[2], int_iter[3]);
	}

	hipMemcpy(dFlag, dev_dFlag, numUnit * sizeof(bool), hipMemcpyDeviceToHost);
}

__global__ void BCfindFlagKernel(char* conFlag, float* BC)
{
	int unitID = blockIdx.x;

	if (conFlag[unitID] != 1) return;

	int tid = threadIdx.x;
	int unitLength = blockDim.x;
	__shared__ float BCbufferMin[MAX_UNIT_LENGTH];
	__shared__ float BCbufferMax[MAX_UNIT_LENGTH];

	BCbufferMin[tid] = BC[unitID * unitLength + tid];
	BCbufferMax[tid] = BC[unitID * unitLength + tid];
	__syncthreads();

	for (int i = 1; i < unitLength; i <<= 1) {
		if (tid % (i << 1) == 0 && tid + i < unitLength) {
			BCbufferMin[tid] = BCbufferMin[tid] < BCbufferMin[tid + i] ? BCbufferMin[tid] : BCbufferMin[tid + i];
			BCbufferMax[tid] = BCbufferMax[tid] > BCbufferMax[tid + i] ? BCbufferMax[tid] : BCbufferMax[tid + i];
		}
		__syncthreads();
	}

	if (tid == 0) {
		// NORMAL INEQUALITY

		if (BCbufferMin[tid] >= 0) {
			conFlag[unitID] = 0;
		}
		else if (BCbufferMax[tid] <= 0) {
			conFlag[unitID] = 2;
		}
		else {
			conFlag[unitID] = 1;
		}

		// BOUNDED INEQUALITY
		/*
		if (BCbufferMin[tid] >= 0 && BCbufferMax[tid] <= 1) {
			conFlag[unitID] = 2;
		}
		else if (BCbufferMin[tid] > 1 || BCbufferMax[tid] < 0) {
			conFlag[unitID] = 0;
		}
		else {
			conFlag[unitID] = 1;
		}
		*/
	}
}

__global__ void BCfindIntFlagKernel(char* intFlag, char* conFlag, uint8_t numCons) {
	int flagID = blockIdx.x;
	bool ifSatisfy = true;
	for (int intID = flagID * numCons; intID < (flagID + 1) * numCons; intID++) {
		if (conFlag[intID] == 0) {
			intFlag[flagID] = 0;
			return;
		}
		else if (conFlag[intID] == 1) {
			ifSatisfy = false;
		}
	}
	intFlag[flagID] = ifSatisfy ? 2 : 1;
}

__global__ void BCfindEquFlagKernel(bool* equFlag, float* BC)
{
	int unitID = blockIdx.x;

	int tid = threadIdx.x;
	int unitLength = blockDim.x;
	__shared__ float BCbufferMin[MAX_UNIT_LENGTH];
	__shared__ float BCbufferMax[MAX_UNIT_LENGTH];

	BCbufferMin[tid] = BC[unitID * unitLength + tid];
	BCbufferMax[tid] = BC[unitID * unitLength + tid];
	__syncthreads();

	for (int i = 1; i < unitLength; i <<= 1) {
		if (tid % (i << 1) == 0 && tid + i < unitLength) {
			BCbufferMin[tid] = BCbufferMin[tid] < BCbufferMin[tid + i] ? BCbufferMin[tid] : BCbufferMin[tid + i];
			BCbufferMax[tid] = BCbufferMax[tid] > BCbufferMax[tid + i] ? BCbufferMax[tid] : BCbufferMax[tid + i];
		}
		__syncthreads();
	}

	if (tid == 0) {
		if (BCbufferMin[tid] > 0 || BCbufferMax[tid] < 0) {
			equFlag[unitID] = 0;
		}
		else {
			equFlag[unitID] = 1;
		}
	}
}

__global__ void BCfindEFlagKernel(bool* eFlag, bool* equsFlag, uint8_t numEqus) {
	int flagID = blockIdx.x;
	for (int intID = flagID * numEqus; intID < (flagID + 1) * numEqus; intID++) {
		if (equsFlag[intID] == 0) {
			eFlag[flagID] = 0;
			return;
		}
	}
	eFlag[flagID] = 1;
}

__global__ void BCfindBoundKernel(float* bdMin, float* bdMax, float* BC)
{
	int unitID = blockIdx.x;
	int tid = threadIdx.x;
	int unitLength = blockDim.x;
	__shared__ float BCbufferMin[MAX_UNIT_LENGTH];
	__shared__ float BCbufferMax[MAX_UNIT_LENGTH];

	BCbufferMin[tid] = BC[unitID * unitLength + tid];
	BCbufferMax[tid] = BC[unitID * unitLength + tid];
	__syncthreads();

	for (int i = 1; i < unitLength; i <<= 1) {
		if (tid % (i << 1) == 0 && tid + i < unitLength) {
			BCbufferMin[tid] = BCbufferMin[tid] < BCbufferMin[tid + i] ? BCbufferMin[tid] : BCbufferMin[tid + i];
			BCbufferMax[tid] = BCbufferMax[tid] > BCbufferMax[tid + i] ? BCbufferMax[tid] : BCbufferMax[tid + i];
		}
		__syncthreads();
	}

	if (tid == 0) {
		bdMin[unitID] = BCbufferMin[tid];
		bdMax[unitID] = BCbufferMax[tid];
	}
}

__global__ void BCfindDerivativeKernel(bool* pdFlag, float* BC, float* pdValue)
{
	int unitID = blockIdx.x;

	if (!pdFlag[unitID]) return;

	int tid = threadIdx.x;
	int unitLength = blockDim.x;
	__shared__ float BCbufferMin[MAX_UNIT_LENGTH];
	__shared__ float BCbufferMax[MAX_UNIT_LENGTH];

	BCbufferMin[tid] = BC[unitID * unitLength + tid];
	BCbufferMax[tid] = BC[unitID * unitLength + tid];
	__syncthreads();

	for (int i = 1; i < unitLength; i <<= 1) {
		if (tid % (i << 1) == 0 && tid + i < unitLength) {
			BCbufferMin[tid] = BCbufferMin[tid] < BCbufferMin[tid + i] ? BCbufferMin[tid] : BCbufferMin[tid + i];
			BCbufferMax[tid] = BCbufferMax[tid] > BCbufferMax[tid + i] ? BCbufferMax[tid] : BCbufferMax[tid + i];
		}
		__syncthreads();
	}

	if (tid == 0) {
		if (BCbufferMin[tid] > pdValue[unitID] || BCbufferMax[tid] < pdValue[unitID]) {
			pdFlag[unitID] = false;
		}
		else {
			pdFlag[unitID] = true;
		}
	}
}

__global__ void biBCfinddFlagKernel(bool* dFlag, bool* pdFlag, uint8_t numDimension, uint32_t* interval, uint8_t iter_0, uint8_t iter_1) {
	int flagID = blockIdx.x;
	for (int pdID = flagID * numDimension; pdID < (flagID + 1) * numDimension; pdID++) {
		if (pdFlag[pdID] == false) {
			int intID = flagID * numDimension;
			if (interval[intID] == 0 || (interval[intID] + 1) == (1 << (uint32_t)iter_0)) {
				dFlag[flagID] = true;
				return;
			}
			if (interval[intID + 1] == 0 || (interval[intID + 1] + 1) == (1 << (uint32_t)iter_1)) {
				dFlag[flagID] = true;
				return;
			}

			dFlag[flagID] = false;
			return;
		}
	}
	dFlag[flagID] = true;
}

__global__ void triBCfinddFlagKernel(bool* dFlag, bool* pdFlag, uint8_t numDimension, uint32_t* interval, uint8_t iter_0, uint8_t iter_1, uint8_t iter_2) {
	int flagID = blockIdx.x;
	for (int pdID = flagID * numDimension; pdID < (flagID + 1) * numDimension; pdID++) {
		if (pdFlag[pdID] == false) {
			int intID = flagID * numDimension;
			if (interval[intID] == 0 || (interval[intID] + 1) == (1 << (uint32_t)iter_0)) {
				dFlag[flagID] = true;
				return;
			}
			if (interval[intID + 1] == 0 || (interval[intID + 1] + 1) == (1 << (uint32_t)iter_1)) {
				dFlag[flagID] = true;
				return;
			}
			if (interval[intID + 2] == 0 || (interval[intID + 2] + 1) == (1 << (uint32_t)iter_2)) {
				dFlag[flagID] = true;
				return;
			}

			dFlag[flagID] = false;
			return;
		}
	}
	dFlag[flagID] = true;
}

__global__ void quadBCfinddFlagKernel(bool* dFlag, bool* pdFlag, uint8_t numDimension, uint32_t* interval, uint8_t iter_0, uint8_t iter_1, uint8_t iter_2, uint8_t iter_3) {
	int flagID = blockIdx.x;
	for (int pdID = flagID * numDimension; pdID < (flagID + 1) * numDimension; pdID++) {
		if (pdFlag[pdID] == false) {
			int intID = flagID * numDimension;
			if (interval[intID] == 0 || (interval[intID] + 1) == (1 << (uint32_t)iter_0)) {
				dFlag[flagID] = true;
				return;
			}
			if (interval[intID + 1] == 0 || (interval[intID + 1] + 1) == (1 << (uint32_t)iter_1)) {
				dFlag[flagID] = true;
				return;
			}
			if (interval[intID + 2] == 0 || (interval[intID + 2] + 1) == (1 << (uint32_t)iter_2)) {
				dFlag[flagID] = true;
				return;
			}
			if (interval[intID + 3] == 0 || (interval[intID + 3] + 1) == (1 << (uint32_t)iter_3)) {
				dFlag[flagID] = true;
				return;
			}

			dFlag[flagID] = false;
			return;
		}
	}
	dFlag[flagID] = true;
}

void BC::eliminate() {
	estiMin = FLT_MAX;
	float estiMinMin = FLT_MAX;
	final_index = 0xffffffff;

	if (numCons > 0 && numEqus > 0) {
		for (uint32_t i = 0; i < numUnit; i++) {
			if (eFlag[i] == 1) {
				if (intFlag[i] == 2 && bdMax[i] < estiMin) {
					estiMin = bdMax[i];
					final_index = i;
				}
				if (intFlag[i] > 0 && bdMin[i] < estiMinMin) {
					estiMinMin = bdMin[i];
				}
			}
		}
	}
	else if (numCons == 0 && numEqus > 0) {
		for (uint32_t i = 0; i < numUnit; i++) {
			if (eFlag[i] == 1) {
				if (bdMax[i] < estiMin) {
					estiMin = bdMax[i];
					final_index = i;
				}
				if (bdMin[i] < estiMinMin) {
					estiMinMin = bdMin[i];
				}
			}
		}
	}
	else if (numCons > 0 && numEqus == 0) {
		for (uint32_t i = 0; i < numUnit; i++) {
			if (intFlag[i] == 2 && bdMax[i] < estiMin) {
				estiMin = bdMax[i];
				final_index = i;
			}
			if (intFlag[i] > 0 && bdMin[i] < estiMinMin) {
				estiMinMin = bdMin[i];
			}
		}
	}
	else {
		for (uint32_t i = 0; i < numUnit; i++) {
			if (bdMax[i] < estiMin) {
				estiMin = bdMax[i];
				final_index = i;
			}
			if (bdMin[i] < estiMinMin) {
				estiMinMin = bdMin[i];
			}
		}
	}

	estimated_accuracy = estiMin - estiMinMin;
	if (estimated_accuracy <= target_accuracy && estiMinMin != FLT_MAX) {
		last = true;
		return;
	}

	if ((numUnit << 1) > MAX_UNIT_NUM) return;

	uint32_t elimNum = 0;
	uint32_t saveNum = 0;
	uint32_t replaceNum;
	if (numCons > 0 && numEqus > 0) {
		for (uint32_t i = 0; i < numUnit; i++) {
			if (intFlag[i] == 0 || eFlag[i] == 0) {
				elimPos[elimNum++] = i;
			}
			else {
				if (bdMin[i] > estiMin) {
					elimPos[elimNum++] = i;
				}
				else {
					/*
					if (dFlag[i] == false && intFlag[i] == 2) {
						elimPos[elimNum++] = i;
					}
					else {
						savePos[saveNum++] = i;
					}
					*/
					savePos[saveNum++] = i;
				}
			}
		}
	}
	else if (numCons == 0 && numEqus > 0) {
		for (uint32_t i = 0; i < numUnit; i++) {
			if (eFlag[i] == 0) {
				elimPos[elimNum++] = i;
			}
			else {
				if (bdMin[i] > estiMin) {
					elimPos[elimNum++] = i;
				}
				else {
					savePos[saveNum++] = i;
				}
			}
		}
	}
	else if (numCons > 0 && numEqus == 0) {
		for (uint32_t i = 0; i < numUnit; i++) {
			if (intFlag[i] == 0) {
				elimPos[elimNum++] = i;
			}
			else {
				if (bdMin[i] > estiMin) {
					elimPos[elimNum++] = i;
				}
				else {
					if (dFlag[i] == false && intFlag[i] == 2) {
						elimPos[elimNum++] = i;
					}
					else {
						savePos[saveNum++] = i;
					}
				}
			}
		}
	}
	else {
		for (uint32_t i = 0; i < numUnit; i++) {
			if (bdMin[i] > estiMin) {
				elimPos[elimNum++] = i;
			}
			else {
				if (dFlag[i] == false) {
					elimPos[elimNum++] = i;
				}
				else {
					savePos[saveNum++] = i;
				}
			}
		}
	}
	
	if (saveNum > 0 && elimNum > 0 && elimPos[0] < saveNum) {
		replaceNum = elimNum;
		for (uint32_t i = 0; i < elimNum; i++) {
			if (elimPos[i] >= saveNum) {
				replaceNum = i;
				break;
			}
		}

		hipMemcpy(dev_elimPos, elimPos, replaceNum * sizeof(uint32_t), hipMemcpyHostToDevice);
		hipMemcpy(dev_savePos, savePos, saveNum * sizeof(uint32_t), hipMemcpyHostToDevice);

		BCeliminateKernelPart1 << < replaceNum, opt_unitLength >> > (dev_opt_BC, dev_elimPos, dev_savePos, saveNum);

		BCeliminateKernelPart2 << < replaceNum, numDimension >> > (dev_interval, dev_elimPos, dev_savePos, saveNum);

		if (numCons > 0) {
			dim3 grid1(replaceNum, numCons, 1);
			BCeliminateKernelPart3 << < grid1, con_unitLength >> > (dev_con_BC, dev_consFlag, dev_elimPos, dev_savePos, saveNum);
		}

		if (numEqus > 0) {
			BCeliminateKernelPart1 << < replaceNum * numEqus, equ_unitLength >> > (dev_equ_BC, dev_elimPos, dev_savePos, saveNum);
		}

		dim3 grid2(replaceNum, numDimension, 1);
		BCeliminateKernelPart4 << < grid2, opt_unitLength >> > (dev_pd_BC, dev_pdFlag, dev_elimPos, dev_savePos, saveNum);
	}

	numUnit = saveNum;
}

__global__ void BCeliminateKernelPart1(float* target_BC, uint32_t* elimPos, uint32_t* savePos, uint32_t saveNum) {
	int unitID = elimPos[blockIdx.x];
	int replaceID = savePos[saveNum - blockIdx.x - 1];
	int BCID = threadIdx.x;
	int unitLength = blockDim.x;
	target_BC[unitID * unitLength + BCID] = target_BC[replaceID * unitLength + BCID];
}

__global__ void BCeliminateKernelPart2(uint32_t* target_interval, uint32_t* elimPos, uint32_t* savePos, uint32_t saveNum) {
	int unitID = elimPos[blockIdx.x];
	int replaceID = savePos[saveNum - blockIdx.x - 1];
	int intID = threadIdx.x;
	int numDimension = blockDim.x;
	target_interval[unitID * numDimension + intID] = target_interval[replaceID * numDimension + intID];
}

__global__ void BCeliminateKernelPart3(float* target_BC, char* target_consFlag, uint32_t* elimPos, uint32_t* savePos, uint32_t saveNum) {
	int unitID = elimPos[blockIdx.x];
	int replaceID = savePos[saveNum - blockIdx.x - 1];
	int conID = blockIdx.y;
	int numCons = gridDim.y;
	int BCID = threadIdx.x;
	int unitLength = blockDim.x;
	target_BC[(unitID * numCons + conID) * unitLength + BCID] = target_BC[(replaceID * numCons + conID) * unitLength + BCID];

	if (BCID == 0) {
		target_consFlag[unitID * numCons + conID] = target_consFlag[replaceID * numCons + conID];
	}
}

__global__ void BCeliminateKernelPart4(float* target_BC, bool* target_pdFlag, uint32_t* elimPos, uint32_t* savePos, uint32_t saveNum) {
	int unitID = elimPos[blockIdx.x];
	int replaceID = savePos[saveNum - blockIdx.x - 1];
	int pdID = blockIdx.y;
	int numDimension = gridDim.y;
	int BCID = threadIdx.x;
	int unitLength = blockDim.x;
	target_BC[(unitID * numDimension + pdID) * unitLength + BCID] = target_BC[(replaceID * numDimension + pdID) * unitLength + BCID];

	if (BCID == 0) {
		target_pdFlag[unitID * numDimension + pdID] = target_pdFlag[replaceID * numDimension + pdID];
	}
}

void BC::finalResult() {
	final_result = new float[numDimension];
	if(final_index == 0xffffffff){
		intervalRes = new float[numUnit * numDimension];
		hipMalloc((void**)&dev_intervalRes, numUnit * numDimension * sizeof(float));
		if (numDimension == 2) {
			biBCfinalResultKernel << < numUnit, numDimension >> > (dev_intervalRes, dev_interval, int_iter[0], int_iter[1]);
		}
		else if (numDimension == 3) {
			triBCfinalResultKernel << < numUnit, numDimension >> > (dev_intervalRes, dev_interval, int_iter[0], int_iter[1], int_iter[2]);
		}
		else {
			quadBCfinalResultKernel << < numUnit, numDimension >> > (dev_intervalRes, dev_interval, int_iter[0], int_iter[1], int_iter[2], int_iter[3]);
		}
		hipMemcpy(intervalRes, dev_intervalRes, numUnit * numDimension * sizeof(float), hipMemcpyDeviceToHost);

		candidates = new float[numUnit];
		estiMin = FLT_MAX;
		for (uint32_t optID = 0; optID < numUnit; optID++) {
			candidates[optID] = 0;
			for (uint16_t k = 0; k < opt->numTerms; k++)
			{
				float result = opt->coeff[k];
				for (uint16_t i = 0; i < numDimension; i++)
				{
					for (uint16_t j = 0; j < opt->degree[k * numDimension + i]; j++)
					{
						result *= intervalRes[optID * numDimension + i];
					}
				}
				candidates[optID] += result;
			}

			if (candidates[optID] < estiMin) {
				estiMin = candidates[optID];
				final_index = optID;
			}
		}

		for(int i = 0; i < numDimension; i++){
			final_result[i] = intervalRes[final_index * numDimension + i];
		}
	}
	else{
		hipMemcpy(interval, dev_interval + final_index * numDimension, numDimension * sizeof(uint32_t), hipMemcpyDeviceToHost);
		for(int i = 0; i < numDimension; i++){
			final_result[i] = ((float)(interval[i]) + 0.5) / (float)(1 << (uint32_t)int_iter[i]);
		}
	}
}

__global__ void biBCfinalResultKernel(float* target_intervalRes, uint32_t* interval, uint8_t iter_1, uint8_t iter_2) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadIdx.x == 0) {
		target_intervalRes[id] = ((float)(interval[id]) + 0.5) / (float)(1 << ((uint32_t)iter_1));
	}
	else {
		target_intervalRes[id] = ((float)(interval[id]) + 0.5) / (float)(1 << ((uint32_t)iter_2));
	}
}

__global__ void triBCfinalResultKernel(float* target_intervalRes, uint32_t* interval, uint8_t iter_1, uint8_t iter_2, uint8_t iter_3) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadIdx.x == 0) {
		target_intervalRes[id] = ((float)(interval[id]) + 0.5) / (float)(1 << ((uint32_t)iter_1));
	}
	else if (threadIdx.x == 1) {
		target_intervalRes[id] = ((float)(interval[id]) + 0.5) / (float)(1 << ((uint32_t)iter_2));
	}
	else {
		target_intervalRes[id] = ((float)(interval[id]) + 0.5) / (float)(1 << ((uint32_t)iter_3));
	}
}

__global__ void quadBCfinalResultKernel(float* target_intervalRes, uint32_t* interval, uint8_t iter_1, uint8_t iter_2, uint8_t iter_3, uint8_t iter_4) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadIdx.x == 0) {
		target_intervalRes[id] = ((float)(interval[id]) + 0.5) / (float)(1 << ((uint32_t)iter_1));
	}
	else if (threadIdx.x == 1) {
		target_intervalRes[id] = ((float)(interval[id]) + 0.5) / (float)(1 << ((uint32_t)iter_2));
	}
	else if (threadIdx.x == 2) {
		target_intervalRes[id] = ((float)(interval[id]) + 0.5) / (float)(1 << ((uint32_t)iter_3));
	}
	else {
		target_intervalRes[id] = ((float)(interval[id]) + 0.5) / (float)(1 << ((uint32_t)iter_4));
	}
}

int BC::solve(bool debugMode, bool verboseMode) {
	int exitFlag = 1; // 1 means everything is fine, -12345 means infeasible, -54321 means too many boxes have been generated before stopping criteria could be satisfied, the current result may be inaccurate
	last = false;

	findFlag();

	target_accuracy = (bdMax[0] - bdMin[0]) * STOPPING_CRITERIA;
	mexPrintf("Target accuracy: %f\n", target_accuracy);

	for (iter = 1; iter <= MAX_ITER_NUM; iter++) {
		bool ifBreak = false;
		for (dim = 0; dim < numDimension; dim++) {
			if ((numUnit << 1) > MAX_UNIT_NUM) {
				mexPrintf("Too many units, the program exits without meeting the stopping criteria!\n");
				exitFlag = -54321;
				ifBreak = true;
				break;
			}

			if (verboseMode) mexPrintf("Start iteration %d dim %d\n", iter, dim);
			dilation(dim);
			apex_numUnit = numUnit > apex_numUnit ? numUnit : apex_numUnit;
			int_iter[dim]++;
			if (verboseMode) mexPrintf("Dilation patch number: %d\n", numUnit);
			findFlag();
			if (debugMode) debug_print();
			eliminate();

			if (verboseMode) mexPrintf("Final patch number: %d\nEstimated Minimum: %.8f\nEstimated Bound: %.8f\n", numUnit, estiMin, estimated_accuracy);
			if (debugMode) debug_print();
			if (verboseMode) mexPrintf("\n");

			if (numUnit == 0) {
				mexPrintf("Infeasible!\n");
				exitFlag = -12345;
				ifBreak = true;
				break;
			}

			if (last) {
				ifBreak = true;
				break;
			}
		}
		
		mexPrintf("Finish Iteration %d\n", iter);
		mexPrintf("Final Patch number: %d\nEstimated Minimum: %.8f\n", numUnit, estiMin);

		if (ifBreak) break;
	}

	mexPrintf("Estimated accuracy: %f\n", estimated_accuracy);

	if (exitFlag != -12345) {
		finalResult();
	}

	return exitFlag;
}

#endif
