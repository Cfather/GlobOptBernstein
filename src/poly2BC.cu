# ifndef POLY_2_BC_CPP_INCLUDED
# define POLY_2_BC_CPP_INCLUDED
# include "poly2BC.h"

poly::poly()
{
	numDimension = 0;
	numTerms = 0;
	coeff = NULL;
	degree = NULL;
	maxDegree = NULL;
}

poly::~poly()
{
	delete[] coeff;
	delete[] degree;
	delete[] maxDegree;
}

poly::poly(uint32_t numDimension_input, uint32_t numTerms_input, double* data_mat)
{
	numDimension = numDimension_input;
	numTerms = numTerms_input;

	coeff = new float[numTerms];
	for (uint32_t i = 0; i < numTerms; i++)
	{
		coeff[i] = (float)data_mat[i * (numDimension + 1) + numDimension];
	}

	degree = new uint32_t[numTerms * numDimension];
	maxDegree = new uint32_t[numDimension];
	for (uint32_t i = 0; i < numDimension; i++)
	{
		maxDegree[i] = 0;
		for (uint32_t j = 0; j < numTerms; j++)
		{
			degree[j * numDimension + i] = (uint32_t)data_mat[j * (numDimension + 1) + i];
			if (degree[j * numDimension + i] > maxDegree[i])
			{
				maxDegree[i] = degree[j * numDimension + i];
			}
		}
		maxDegree[i] += 1;
	}
}

poly::poly(uint32_t numDimension_input, uint32_t numTerms_input, float* data_mat)
{
	numDimension = numDimension_input;
	numTerms = numTerms_input;

	coeff = new float[numTerms];
	for (uint32_t i = 0; i < numTerms; i++)
	{
		coeff[i] = data_mat[i * (numDimension + 1) + numDimension];
	}

	degree = new uint32_t[numTerms * numDimension];
	maxDegree = new uint32_t[numDimension];
	for (uint32_t i = 0; i < numDimension; i++)
	{
		maxDegree[i] = 0;
		for (uint32_t j = 0; j < numTerms; j++)
		{
			degree[j * numDimension + i] = (uint32_t)data_mat[j * (numDimension + 1) + i];
			if (degree[j * numDimension + i] > maxDegree[i])
			{
				maxDegree[i] = degree[j * numDimension + i];
			}
		}
		maxDegree[i] += 1;
	}
}

poly::poly(uint32_t numDimension_input, uint32_t numTerms_input, double* degree_input, double* coef_input) {
	numDimension = numDimension_input;
	numTerms = numTerms_input;

	coeff = new float[numTerms];
	for (uint32_t i = 0; i < numTerms; i++)
	{
		coeff[i] = (float)coef_input[i];
	}

	degree = new uint32_t[numTerms * numDimension];
	maxDegree = new uint32_t[numDimension];
	for (uint32_t i = 0; i < numDimension; i++)
	{
		maxDegree[i] = 0;
		for (uint32_t j = 0; j < numTerms; j++)
		{
			degree[j * numDimension + i] = (uint32_t)degree_input[j * numDimension + i];
			if (degree[j * numDimension + i] > maxDegree[i])
			{
				maxDegree[i] = degree[j * numDimension + i];
			}
		}
		maxDegree[i] += 1;
	}
}

void poly::printDetails() {
	mexPrintf("DIMENSION: %d, TERMS: %d\n", numDimension, numTerms);
	for (uint32_t i = 0; i < numTerms; i++) {
		mexPrintf("%.6f,", coeff[i]);
		for (uint32_t j = 0; j < numDimension; j++) {
			mexPrintf(" %d,", degree[i * numDimension + j]);
		}
		mexPrintf("\n");
	}
}

void poly::partialDerivative(poly* &res, float &res_value, uint32_t dim) {
	float* data_mat = new float[numTerms * (numDimension + 1)];
	uint32_t pd_numTerms = 0;
	float pd_value = 0;
	for (uint32_t i = 0; i < numTerms; i++) {
		if (degree[i * numDimension + dim] > 1) {
			data_mat[pd_numTerms * (numDimension + 1) + numDimension] = coeff[i] * degree[i * numDimension + dim];

			for (uint32_t j = 0; j < numDimension; j++) {
				data_mat[pd_numTerms * (numDimension + 1) + j] = degree[i * numDimension + j];
			}
			data_mat[pd_numTerms * (numDimension + 1) + dim]--;

			pd_numTerms++;
		}
		else if (degree[i * numDimension + dim] == 1) {
			uint32_t temp = 0;
			for (uint32_t j = 0; j < numDimension; j++) {
				if (j != dim) {
					temp += degree[i * numDimension + j];
				}
			}

			if (temp > 0) {
				data_mat[pd_numTerms * (numDimension + 1) + numDimension] = coeff[i] * degree[i * numDimension + dim];

				for (uint32_t j = 0; j < numDimension; j++) {
					data_mat[pd_numTerms * (numDimension + 1) + j] = degree[i * numDimension + j];
				}
				data_mat[pd_numTerms * (numDimension + 1) + dim]--;

				pd_numTerms++;
			}
			else {
				pd_value -= coeff[i];
			}
		}
	}

	res = new poly(numDimension, pd_numTerms, data_mat);
	res_value = pd_value;

	delete[] data_mat;
}

# endif